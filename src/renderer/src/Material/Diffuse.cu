#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/RayType.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/Helpers/random.h"
#include "Renderer/Helpers/helpers.h"
#include "Renderer/Helpers/samplers.h"
#include "Renderer/Helpers/store_photon.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(float3, Kd, , );

/*
// Radiance Program
*/

RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    float3 value3 = Kd;
    //value3 = make_float3(value3.x+0.5, value3.y+0.5, value3.z+0.5);
    //value3 /= 1.5;
    //value3.x = powf(value3.x,2.4);
    //value3.y = powf(value3.y,2.4);
    //value3.z = powf(value3.z,2.4);
    radiancePrd.attenuation *= value3;
    //radiancePrd.attenuation *= Kd;
    radiancePrd.normal = worldShadingNormal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
}

/*
// Photon Program
*/

RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
    {
        Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
        STORE_PHOTON(photon);
    }

    photonPrd.power *= Kd;
    OPTIX_DEBUG_PRINT(photonPrd.depth, "Hit Diffuse P(%.2f %.2f %.2f) RT=%d\n", hitPoint.x, hitPoint.y, hitPoint.z, ray.ray_type);
    photonPrd.weight *= fmaxf(Kd);

    // Use russian roulette sampling from depth X to limit the length of the path

    if( photonPrd.depth >= PHOTON_TRACING_RR_START_DEPTH)
    {
        float probContinue = favgf(Kd);
        float probSample = getRandomUniformFloat(&photonPrd.randomState);
        if(probSample >= probContinue )
        {
            return;
        }
        photonPrd.power /= probContinue;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001)
    {
        return;
    }

    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;

    newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    //newPhotonDirection = sampleUnitHemisphere(worldShadingNormal,getRandomUniformFloat2(&photonPrd.randomState));
    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.01 );
    rtTrace(sceneRootObject, newRay, photonPrd);
}
