#include "hip/hip_runtime.h"
/*
 * This file is part of SceneNet RGB-D.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/scenenet-rgbd-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <Util/helpers.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Renderer/Helpers/random.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/RayType.h"
#include "rendererConfig.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

// Globally set for the whole program
rtDeclareVariable(float, MaxDepth, , );
rtDeclareVariable(unsigned int, MaxDepthInUnits, , );
rtDeclareVariable(unsigned int, InstanceID, , );

RT_PROGRAM void closestHitProgram()
{
    if (tHit < MaxDepth) {
        radiancePrd.gt_depth = static_cast<unsigned int>(MaxDepthInUnits * (tHit / MaxDepth));
    } else {
        radiancePrd.gt_depth = 0;
    }
    radiancePrd.instance_id = InstanceID;
}
