#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/RayType.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Renderer/PhotonMapping/Photon.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(float3, Kr, , );


RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    radiancePrd.depth++;
    if(radiancePrd.depth <= MAX_RADIANCE_TRACE_DEPTH)
    {
        radiancePrd.attenuation *= Kr;
        float3 newRayDirection = reflect(ray.direction, worldShadingNormal);
        Ray newRay ( hitPoint, newRayDirection, RayType::RADIANCE, 0.01, RT_DEFAULT_MAX );
        rtTrace( sceneRootObject, newRay, radiancePrd );
    }
    radiancePrd.lastTHit = tHit;
}

RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    photonPrd.depth++;
    if (photonPrd.depth <= MAX_PHOTON_TRACE_DEPTH)
    {
        photonPrd.power *= Kr;
        float3 newPhotonDirection = reflect(ray.direction, worldShadingNormal);
        Ray newPhoton (hitPoint, newPhotonDirection, RayType::PHOTON, 0.01 );
        rtTrace(sceneRootObject, newPhoton, photonPrd);
    }
}
