#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

//#define MAX_DEPTH 20

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/Light.h"
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/PhotonMapping/PhotonGrid.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/RayType.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtBuffer<float3, 2> indirectRadianceBuffer;

rtDeclareVariable(float, alpha, , );
rtDeclareVariable(float, emittedPhotonsPerIterationFloat, , );
rtDeclareVariable(float, maxPhotonIterations, , );
rtDeclareVariable(float, ppmRadius, ,);
rtDeclareVariable(float, ppmRadiusSquared, ,);
rtDeclareVariable(float, ppmRadiusSquaredNew, ,);
rtDeclareVariable(float, numPhotonMaps, , );
rtDeclareVariable(uint, localIterationNumber, , );

rtBuffer<Photon, 1> photonKdTree;

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<uint, 2> debugIndirectRadianceCellsVisisted;
rtBuffer<uint, 2> debugIndirectRadiancePhotonsVisisted;
#endif

__device__ __inline float validPhoton(const Photon & photon, const float distance2, const float radius2, const float3 & hitNormal)
{
    /*
    float dot = hitNormal.x * -photon.rayDirection[0];
    dot += hitNormal.y * -photon.rayDirection[1];
    dot += hitNormal.z * -photon.rayDirection[2];
    return distance2 <= radius2;// && dot >= 0; 
    */
    return distance2 <= radius2 && dot(-photon.rayDirection, hitNormal) >= 0; 
}

__device__ __inline float3 photonPower(const Photon & photon, const float distance2, const float radius2)
{
    // Use the gaussian filter from Realistic Image Synthesis Using Photon Mapping, Wann Jensen
    const float alpha = 1.818;
    const float beta = 1.953;
    const float expNegativeBeta = 0.141847;
    float weight = alpha*(1 - (1-exp(-beta*distance2/(2*radius2)))/(1-expNegativeBeta));
    /*
    float3 photonPower;
    photonPower.x = static_cast<float>(photon.power[0]) / 255000.0;
    photonPower.y = static_cast<float>(photon.power[1]) / 255000.0;
    photonPower.z = static_cast<float>(photon.power[2]) / 255000.0;
    */
    return photon.power*weight;
}

RT_PROGRAM void kernel()
{
    clock_t start = clock();
    Hitpoint rec = raytracePassOutputBuffer[launchIndex];
    
    float3 indirectAccumulatedPower = make_float3( 0.0f, 0.0f, 0.0f );

    int _dPhotonsVisited = 0;

    //if (false)
    if(rec.flags & PRD_HIT_NON_SPECULAR)
    {
        float radius2 = ppmRadiusSquared;

        // This code is based on the PPM sample in Optix 3.0.0 SDK by NVIDIA

        const size_t MAX_DEPTH = 21;
        unsigned int stack[MAX_DEPTH];
        unsigned int stack_current = 0;
        unsigned int node = 0;
        #define push_node(N) stack[stack_current++] = (N)
        #define pop_node() stack[--stack_current]

        push_node(0);
        do 
        {
            Photon& photon = photonKdTree[ node ];
            _dPhotonsVisited++;
            uint axis = photon.axis;
            if( !( axis & PPM_NULL ) )
            {
                float3 diff = rec.position - photon.position;
                float distance2 = dot(diff, diff);
                if(validPhoton(photon, distance2, radius2, rec.normal))
                {
                    indirectAccumulatedPower += photonPower(photon, distance2, radius2);
                }

                // Recurse
                if( !( axis & PPM_LEAF ) ) {
                    float d;
                    if      ( axis & PPM_X ) d = diff.x;
                    else if ( axis & PPM_Y ) d = diff.y;
                    else                     d = diff.z;
                    // Calculate the next child selector. 0 is left, 1 is right.
                    int selector = d < 0.0f ? 0 : 1;
                    if( d*d < radius2 ) {
                        push_node( (node<<1) + 2 - selector );
                    }
                    node = (node<<1) + 1 + selector;
                } else {
                    node = pop_node();
                }
            } else {
                node = pop_node();
            }
        }
        while ( node && stack_current < MAX_DEPTH );
    }

    float3 indirectRadiance = indirectAccumulatedPower * rec.attenuation * (1.0f/(M_PIf*ppmRadiusSquared)) *  (1.0f/(maxPhotonIterations*emittedPhotonsPerIterationFloat));

    // Add contribution from volumetric radiance
#if ENABLE_PARTICIPATING_MEDIA
    indirectRadiance += rec.volumetricRadiance / emittedPhotonsPerIterationFloat;
#endif

    if (localIterationNumber == 0) {
        indirectRadianceBuffer[launchIndex] = indirectRadiance;
    } else {
        indirectRadianceBuffer[launchIndex] += indirectRadiance;
    }

#if ENABLE_RENDER_DEBUG_OUTPUT
    debugIndirectRadianceCellsVisisted[launchIndex] = _dCellsVisited;
    debugIndirectRadiancePhotonsVisisted[launchIndex] = _dPhotonsVisited;
#endif

}
