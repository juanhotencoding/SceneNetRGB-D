#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "rendererConfig.h"
#include "Renderer/Light.h"
#include "Renderer/ShadowPRD.h"
#include "Renderer/RayType.h"
#include "Renderer/Helpers/helpers.h"
#include "Renderer/Helpers/samplers.h"
#include "Renderer/Helpers/random.h"
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Math/Sphere.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Photon, 1> photons;
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(uint, photonLaunchWidth, , );
rtDeclareVariable(uint, photonLaunchHeight, , );
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

rtDeclareVariable(uint, photonIterationNumber, , );

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

static __device__ void generatePhotonOriginAndDirection(const Light& light, RandomState& state, const Sphere & boundingSphere, 
    float3& origin, float3& direction, float& photonPowerFactor)
{
    origin = light.position;
    float2 sample1 = getRandomUniformFloat2(&state);

    if(light.lightType == Light::POLYAREA)
    {
        float2 sample2 = getRandomUniformFloat2(&state);
        float total = sample2.x+sample2.y;
        sample2.x /= total;
        sample2.y /= total;
        origin += sample1.x*(optix::float3)light.v1 + sample1.y*(optix::float3)light.v2;
        direction = sampleUnitHemisphere(light.normal, sample2);
    }
    else if(light.lightType == Light::AREA)
    {
        float2 sample2 = getRandomUniformFloat2(&state);
        origin += sample1.x*(optix::float3)light.v1 + sample1.y*(optix::float3)light.v2;
        direction = sampleUnitHemisphere(light.normal, sample2);
        //direction = sampleUnitSphere(sample1);
    }
    else if(light.lightType == Light::POINT)
    {
        // If the point light is well outside the bounding sphere, we make sure to emit 
        // only at the scene (to emulate a directional light)
        float3 sceneCenterToLight = light.position-boundingSphere.center;
        float lightDistance = length(sceneCenterToLight);
        sceneCenterToLight /= lightDistance;
        bool lightWellOutsideSphere = (lightDistance > 1.5*boundingSphere.radius);
        // If light is far away, send photons at the scene and reduce the power based on the solid angle of the scene bounding sphere
        if(lightWellOutsideSphere)
        {
            float3 pointOnDisc = sampleDisc(sample1, boundingSphere.center, boundingSphere.radius, sceneCenterToLight);
            direction = normalize(pointOnDisc-origin);
            // Solid angle of sample disc calculated with http://planetmath.org/calculatingthesolidangleofdisc
            photonPowerFactor = (1  - lightDistance * rsqrtf(boundingSphere.radius*boundingSphere.radius+lightDistance*lightDistance)) / 2.f;
        }
        else
        {
            direction = sampleUnitSphere(sample1);
            origin += direction * light.radius;
        }
    }
    else if(light.lightType == Light::SPOT)
    {
        float3 pointOnDisc = sampleDisc(sample1, origin+light.direction, sinf(light.angle/2), light.direction);
        direction = normalize(pointOnDisc-origin);
    }
}

RT_PROGRAM void generator()
{
    const int num_attempts = 8;
    for (int i = 0; i < num_attempts; ++i) {
        PhotonPRD photonPrd;
        photonPrd.pm_index = ((photonIterationNumber * photonLaunchHeight + launchIndex.y) * photonLaunchWidth + launchIndex.x)*maxPhotonDepositsPerEmitted;
        photonPrd.numStoredPhotons = 0;
        photonPrd.depth = 0;
        photonPrd.weight = 1.0f;
        photonPrd.randomState = randomStates[launchIndex];
        if (photonIterationNumber == 0) {
            for(unsigned int j = 0; j < maxPhotonDepositsPerEmitted; ++j) {
                photons[photonPrd.pm_index+j].power = make_float3(0.0f);
                photons[photonPrd.pm_index+j].position = make_float3(0.0f);
            }
        }

        int lightIndex = 0;
        if(lights.size() > 1)
        {
            float sample = getRandomUniformFloat(&photonPrd.randomState);
            lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
        }

        Light light = lights[lightIndex];
        float powerScale = lights.size();

        photonPrd.power = light.power*powerScale;

        float3 rayOrigin, rayDirection;

        float photonPowerFactor = 1.f;
        generatePhotonOriginAndDirection(light, photonPrd.randomState, sceneBoundingSphere, rayOrigin, rayDirection, photonPowerFactor);
        photonPrd.power *= photonPowerFactor;

        Ray photon = Ray(rayOrigin, rayDirection, RayType::PHOTON, 0.01, RT_DEFAULT_MAX );

        rtTrace( sceneRootObject, photon, photonPrd );

        // Clear photons owned by this thread
        bool waste_photon = false;
        for(unsigned int j = 0; j < maxPhotonDepositsPerEmitted; ++j)
        {
            if(!(fmaxf(photons[photonPrd.pm_index+j].power) > 0.0f)) {
                waste_photon = true;
            }
        }
        randomStates[launchIndex] = photonPrd.randomState;
        if (!waste_photon) {
            break;
        }
    }


#if ENABLE_RENDER_DEBUG_OUTPUT
    debugPhotonPathLengthBuffer[launchIndex] = photonPrd.depth;
#endif

}

rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
RT_PROGRAM void miss()
{
    OPTIX_DEBUG_PRINT(photonPrd.depth, "Photon missed geometry.\n");
}

//
// Exception handler program
//

rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    const unsigned int code = rtGetExceptionCode();

//    RT_EXCEPTION_PROGRAM_ID_INVALID
//    RT_EXCEPTION_TEXTURE_ID_INVALID
//    RT_EXCEPTION_BUFFER_ID_INVALID
//    RT_EXCEPTION_INDEX_OUT_OF_BOUNDS
//    RT_EXCEPTION_STACK_OVERFLOW
//    RT_EXCEPTION_BUFFER_INDEX_OUT_OF_BOUNDS
//    RT_EXCEPTION_INVALID_RAY
//    RT_EXCEPTION_INTERNAL_ERROR
//    RT_EXCEPTION_USER

//    RT_EXCEPTION_ALL
    if( code == RT_EXCEPTION_PROGRAM_ID_INVALID )
    {
        printf("Program Id is invalid\n");
    }
    if( code == RT_EXCEPTION_TEXTURE_ID_INVALID )
    {
        printf("Texture Id is invalid\n");
    }
    if( code == RT_EXCEPTION_BUFFER_ID_INVALID )
    {
        printf("Buffer Id is invalid\n");
    }
    if( code == RT_EXCEPTION_INDEX_OUT_OF_BOUNDS )
    {
        printf("RT_EXCEPTION_INDEX_OUT_OF_BOUNDS\n");
    }
    if( code == RT_EXCEPTION_STACK_OVERFLOW )
    {
        printf("RT_EXCEPTION_STACK_OVERFLOW\n");
    }
    if( code == RT_EXCEPTION_BUFFER_INDEX_OUT_OF_BOUNDS )
    {
        printf("RT_EXCEPTION_BUFFER_INDEX_OUT_OF_BOUNDS\n");
    }
    if( code == RT_EXCEPTION_INVALID_RAY )
    {
        printf("RT_EXCEPTION_INVALID_RAY\n");
    }
    if( code == RT_EXCEPTION_INTERNAL_ERROR )
    {
        printf("RT_EXCEPTION_INTERNAL_ERROR\n");
    }
//    else
//    {
//        printf("Exception User\n");
//    }

//    output_buffer[launch_index] = error;
//    else

    rtPrintExceptionDetails();
    printf("Exception Photon!\n");
    //photonPrd.power = make_float3(0,0,0);
}
