#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Renderer/RadiancePRD.h"
#include "Renderer/ShadowPRD.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Renderer/RayType.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(float3, powerPerArea, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );

/*
// Radiance Program
*/

RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    if (dot(worldShadingNormal,ray.direction) <= 0) {
    float3 Le = powerPerArea/M_PIf;
    radiancePrd.radiance += radiancePrd.attenuation*Le;
    radiancePrd.flags |= PRD_HIT_EMITTER;
    radiancePrd.lastTHit = tHit;
    } else {
    float3 Le = powerPerArea/M_PIf;
    radiancePrd.radiance += 0.01 * radiancePrd.attenuation*Le;
    radiancePrd.flags |= PRD_HIT_EMITTER;
    radiancePrd.lastTHit = tHit;
    }
}

/*
// Photon Program
*/

RT_PROGRAM void closestHitPhoton()
{
   photonPrd.depth++;
    /*
    float3 hitPoint = ray.origin + tHit*ray.direction;
    Ray newPhoton (hitPoint, ray.direction, RayType::PHOTON, 0.01 );
    rtTrace(sceneRootObject, newPhoton, photonPrd);
    */
}

RT_PROGRAM void gatherAnyHitOnEmitter()
{
    shadowPrd.attenuation = 1.0f;
    rtTerminateRay();
}
