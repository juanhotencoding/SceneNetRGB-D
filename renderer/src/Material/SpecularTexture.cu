#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/RayType.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/ShadowPRD.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/Helpers/random.h"
#include "Renderer/Helpers/helpers.h"
#include "Renderer/Helpers/samplers.h"
#include "Renderer/Helpers/store_photon.h"
#include "Renderer/Light.h"

//#define OPTIX_MATERIAL_DUMP

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtBuffer<Light, 1> lights;

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 
rtDeclareVariable(float3, tangent, attribute tangent, ); 
rtDeclareVariable(float3, bitangent, attribute bitangent, ); 
rtDeclareVariable(float2, textureCoordinate, attribute textureCoordinate, );

rtBuffer<Photon, 1> photons;
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> diffuseSampler;
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> normalMapSampler;
rtDeclareVariable(uint, hasNormals, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, alpha, , );

// Radiance Program

__inline__ __device__ float3 getNormalMappedNormal(const float3 & normal,
                                                   const float3 & tangent,
                                                   const float3 & bitangent,
                                                   const float4 & normalMap)
{
    float4 nMap = 2*normalMap - 1;
    float3 N;
    N.x = nMap.x*tangent.x + nMap.y*bitangent.x + nMap.z*normal.x;
    N.y = nMap.x*tangent.y + nMap.y*bitangent.y + nMap.z*normal.y;
    N.z = nMap.x*tangent.z + nMap.y*bitangent.z + nMap.z*normal.z;
    return normalize(N);
}


/* THIS IS CHEAP PHONG SPECULAR - assumes lights are only source of specular
 */
RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 worldGeometryNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
    worldShadingNormal = worldGeometryNormal;
    float3 hitPoint = ray.origin + tHit*ray.direction;

    float3 normal = worldShadingNormal;

    if(hasNormals)
    {
        float3 worldTangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));
        float3 worldBitangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent));
        normal = getNormalMappedNormal(worldShadingNormal, worldTangent, worldBitangent,
                                       tex2D(normalMapSampler, textureCoordinate.x, textureCoordinate.y));
    }


    int numLights = lights.size();
    for(int light_idx = 0; light_idx < numLights; light_idx++) {
        Light & light = lights[light_idx];
        float3 light_pos = light.position;
        float3 light_power = light.power;
        float3 ffnormal = faceforward(worldShadingNormal,-ray.direction,worldGeometryNormal);
        if(light.lightType == Light::AREA)
        {
            const int num_samples = 16;
            float3 stored_radiance = optix::make_float3(0.0);
            for (int sample = 0; sample < num_samples; ++sample) {
                float2 sample = getRandomUniformFloat2(&radiancePrd.randomState);
                float3 light_pos_sample = light_pos + sample.x*light.v1 + sample.y*light.v2;
                float3 l = normalize(light_pos_sample - hitPoint);
                float lightDistance = optix::length(light_pos_sample - hitPoint) + optix::length(tHit*ray.direction);
                float3 n = normalize(normal);
                float3 r = normalize(reflect(l,n));
                float3 v = normalize((ray.origin-hitPoint));
                float dot_val = dot(r,v);
                if (dot_val < 0.0) {
                    ShadowPRD shadowPrd;
                    shadowPrd.attenuation = 1.0f;
                    optix::Ray shadow_ray (hitPoint, l, RayType::SHADOW, 0.05,0.01);
                    rtTrace(sceneRootObject, shadow_ray, shadowPrd);
                    stored_radiance += (shadowPrd.attenuation * Ks * light_power * pow(-dot_val, alpha)) * (1.0 / 500.0 * M_PIf*lightDistance*lightDistance);
                }
            }
            stored_radiance /= num_samples;
            radiancePrd.radiance += stored_radiance;
        }
        else if(light.lightType == Light::POINT)
        {
            float3 l = normalize(light_pos - hitPoint);
            float lightDistance = optix::length(light_pos - hitPoint) + optix::length(tHit*ray.direction);
            float3 n = normalize(normal);
            float3 r = normalize(reflect(l,n));
            float3 v = normalize((ray.origin-hitPoint));
            float dot_val = dot(r,v);
            if (dot_val < 0.0) {
                ShadowPRD shadowPrd;
                shadowPrd.attenuation = 1.0f;
                optix::Ray shadow_ray (hitPoint, l, RayType::SHADOW, 0.01,0.01);
                rtTrace(sceneRootObject, shadow_ray, shadowPrd);
                radiancePrd.radiance += (shadowPrd.attenuation * Ks * light_power * pow(-dot_val, alpha)) * (1.0 / 500.0 * M_PIf*lightDistance*lightDistance);
            }
        }
    }

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.normal = normal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;

    float4 value = tex2D( diffuseSampler, textureCoordinate.x, textureCoordinate.y );
    float3 value3 = make_float3(value.x, value.y, value.z);
    //float3 value3 = make_float3(value.x+0.5, value.y+0.5, value.z+0.5);
    //value3 /= 1.5;
    // Cancel out the standard Camera Response Function which makes colours
    // bleed out
    //value3.x = powf(value3.x,2.4);
    //value3.y = powf(value3.y,2.4);
    //value3.z = powf(value3.z,2.4);
    radiancePrd.attenuation *= value3;
}


/* THIS IS MORE CORRECT BUT MUCH MORE EXPENSIVE
 *
RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 hitPoint = ray.origin + tHit*ray.direction;

    float3 normal = worldShadingNormal;

    if(hasNormals)
    {
        float3 worldTangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));
        float3 worldBitangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent));
        normal = getNormalMappedNormal(worldShadingNormal, worldTangent, worldBitangent,
                                       tex2D(normalMapSampler, textureCoordinate.x, textureCoordinate.y));
    }

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.flags |= PRD_HIT_SPECULAR;
    radiancePrd.normal = normal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
    radiancePrd.depth++;

//    if(radiancePrd.flags & PRD_PATH_TRACING)
//    {
//        radiancePrd.randomNewDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&radiancePrd.randomState));
//    }

    float4 value = tex2D( diffuseSampler, textureCoordinate.x, textureCoordinate.y );
    float3 value3 = make_float3(value.x, value.y, value.z);
    value3.x = powf(value3.x,2.4);
    value3.y = powf(value3.y,2.4);
    value3.z = powf(value3.z,2.4);
    radiancePrd.attenuation *= value3;
    // Add specular component
    float3 reflectedRayDirection = reflect(ray.direction, normal);
    if(radiancePrd.depth <= MAX_SPECULAR_TRACE_DEPTH) {
        float3 cum_radiance = make_float3(0.f);
        for (int i = 0; i < NUM_SPECULAR_SAMPLES; i++) {
            float3 newRayDirection = sampleUnitHemisphereCosExp(reflectedRayDirection, 
                    getRandomUniformFloat2(&radiancePrd.randomState),alpha);
            RadiancePRD subradiance;
            subradiance.attenuation = make_float3( 1.0f );
            subradiance.radiance = make_float3(0.f);
            subradiance.depth = radiancePrd.depth;
            subradiance.flags = 0;
            subradiance.randomState = radiancePrd.randomState;
            Ray newRay = Ray(hitPoint, newRayDirection, RayType::RADIANCE, 0.001, RT_DEFAULT_MAX );
            rtTrace( sceneRootObject, newRay, subradiance);
            radiancePrd.randomState = subradiance.randomState;
            cum_radiance += subradiance.radiance;
        }
        radiancePrd.radiance = (cum_radiance * Ks) / NUM_SPECULAR_SAMPLES;
    }
}
*/

// Photon Program
RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal));
    float3 normal = worldShadingNormal;

    if(hasNormals)
    {
        float3 worldTangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));
        float3 worldBitangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent));
        normal = getNormalMappedNormal(worldShadingNormal, worldTangent, worldBitangent,
                        tex2D(normalMapSampler, textureCoordinate.x, textureCoordinate.y));
    }

    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    /// Record hit if it has bounced at least once
    if(photonPrd.depth >= 1)
    {
        Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
        STORE_PHOTON(photon);
    }

    float4 value = tex2D(diffuseSampler, textureCoordinate.x, textureCoordinate.y);
    float3 value3 = make_float3(value.x, value.y, value.z);
    photonPrd.power *= value3;

#ifdef OPTIX_MATERIAL_DUMP
    for(int i = 0; i<photonPrd.depth;i++) printf("\t");
        printf("Hit diffuse at P(%.2f %.2f %.2f) t=%.3f\n", hitPoint.x, hitPoint.y, hitPoint.z, tHit);
#endif

    photonPrd.weight *= fmaxf(value3);

    // Use russian roulette sampling from depth X to limit the length of the path

    if( photonPrd.depth >= PHOTON_TRACING_RR_START_DEPTH)
    {
        float probContinue = favgf(value3);
        float probSample = getRandomUniformFloat(&photonPrd.randomState);
        if(probSample >= probContinue )
        {
            return;
        }
        photonPrd.power /= probContinue;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.01)
    {
        return;
    }

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU
    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;
#endif

    newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.01 );
    rtTrace(sceneRootObject, newRay, photonPrd);
    /*
    float3 worldShadingNormal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal));
    float3 normal = worldShadingNormal;

    if(hasNormals)
    {
        float3 worldTangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));
        float3 worldBitangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent));
        normal = getNormalMappedNormal(worldShadingNormal, worldTangent, worldBitangent,
                        tex2D(normalMapSampler, textureCoordinate.x, textureCoordinate.y));
    }

    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    float4 value = tex2D(diffuseSampler, textureCoordinate.x, textureCoordinate.y);
    float3 Kd = make_float3(value.x, value.y, value.z);

    // Russian roulette sampling
    float specularDiffuseOrAbsorb = getRandomUniformFloat(&photonPrd.randomState);
    float3 both = Kd + Ks;
    float diffuseThreshold = dot(photonPrd.power,Kd) / dot(photonPrd.power,both);
    float specularThreshold = dot(photonPrd.power,Ks) / dot(photonPrd.power,both);
    if (specularDiffuseOrAbsorb < diffuseThreshold) {
        if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted) {
            Photon photon (photonPrd.power, hitPoint, ray.direction, normal);
            STORE_PHOTON(photon);
        }
        photonPrd.power *= Kd;
        newPhotonDirection = sampleUnitHemisphereCos(normal, getRandomUniformFloat2(&photonPrd.randomState));
    } else if (specularDiffuseOrAbsorb < (diffuseThreshold + specularThreshold)) {
        photonPrd.power *= (Ks/specularThreshold);
        newPhotonDirection = reflect(ray.direction, normal);
    } else {
        return;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001) {
        return;
    }

    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;

    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.01 );
    rtTrace(sceneRootObject, newRay, photonPrd);
    */
}

