#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/RayType.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/PhotonMapping/PhotonPRD.h"
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/Helpers/random.h"
#include "Renderer/Helpers/helpers.h"
#include "Renderer/Helpers/samplers.h"
#include "Renderer/Helpers/store_photon.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
// Possibly needed if we swap to phong or something simpler
//rtBuffer<Light, 1> lights;
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, alpha, , );

__device__ __inline float sumVector(const float3 & vect) {
    return vect.x + vect.y + vect.z;
}


/*
// Radiance Program
*/
RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.flags |= PRD_HIT_SPECULAR;
    radiancePrd.normal = worldShadingNormal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
    radiancePrd.depth++;
    // Add diffuse component
    float3 value3 = Kd;
    //value3.x = powf(value3.x,2.4);
    //value3.y = powf(value3.y,2.4);
    //value3.z = powf(value3.z,2.4);
    radiancePrd.attenuation *= value3;
    //radiancePrd.attenuation *= Kd;
    // Add specular component
    float3 reflectedRayDirection = reflect(ray.direction, worldShadingNormal);
    if(radiancePrd.depth <= MAX_SPECULAR_TRACE_DEPTH) {
        float3 cum_radiance = make_float3(0.f);
        for (int i = 0; i < NUM_SPECULAR_SAMPLES; i++) {
            float3 newRayDirection = sampleUnitHemisphereCosExp(reflectedRayDirection, 
                    getRandomUniformFloat2(&radiancePrd.randomState),alpha);
            RadiancePRD subradiance;
            subradiance.attenuation = make_float3( 1.0f );
            subradiance.radiance = make_float3(0.f);
            subradiance.depth = radiancePrd.depth;
            subradiance.flags = 0;
            subradiance.randomState = radiancePrd.randomState;
            Ray newRay = Ray(hitPoint, newRayDirection, RayType::RADIANCE, 0.001, RT_DEFAULT_MAX );
            rtTrace( sceneRootObject, newRay, subradiance);
            radiancePrd.randomState = subradiance.randomState;
            cum_radiance += subradiance.radiance;
        }
        radiancePrd.radiance = (cum_radiance * Ks) / NUM_SPECULAR_SAMPLES;
    }
}


/*
// Photon Program
*/

RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    // Russian roulette sampling
    float specularDiffuseOrAbsorb = getRandomUniformFloat(&photonPrd.randomState);
    float3 both = Kd + Ks;
    float diffuseThreshold = dot(photonPrd.power,Kd) / dot(photonPrd.power,both);
    float specularThreshold = dot(photonPrd.power,Ks) / dot(photonPrd.power,both);
    if (specularDiffuseOrAbsorb < diffuseThreshold) {
        if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted) {
            Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
            STORE_PHOTON(photon);
        }
        photonPrd.power *= (Kd/diffuseThreshold);
        newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    } else if (specularDiffuseOrAbsorb < (diffuseThreshold + specularThreshold)) {
        photonPrd.power *= (Ks/specularThreshold);
        newPhotonDirection = reflect(ray.direction, worldShadingNormal);
    } else /* ABSORBED */ {
        return;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001) {
        return;
    }

    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;

    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.01 );
    rtTrace(sceneRootObject, newRay, photonPrd);
}
