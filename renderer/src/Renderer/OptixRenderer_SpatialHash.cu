#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>

#include "rendererConfig.h"

#include <optix_world.h>
#include "Renderer/PhotonMapping/Photon.h"
#include "Renderer/PhotonMapping/PhotonGrid.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/OptixRenderer.h"
//#include "Util/sutil/sutil.h"
#include "Renderer/OptixEntryPoint.h"
#include "Renderer/Helpers/optix.h"
#include "Renderer/Helpers/random.h"
#include "Math/Vector3.h"

using namespace optix;

/*
// Initialize random state buffer
*/

static void __global__ initRandomStateBuffer(RandomState* states, unsigned int seed, unsigned int num)
{
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index < num)
    {
        initializeRandomState(&states[index], seed, index);
    }
}

static void initializeRandomStateBuffer(optix::Buffer & buffer, int numStates)
{
    unsigned int seed = 574133*(unsigned int)clock() + 47844152748*(unsigned int)time(NULL);
    printf("Seeding on %d clock: %d time: %d \n", seed, (unsigned int)clock(), (unsigned int)time(NULL));

    RandomState* states = getDevicePtr<RandomState>(buffer, 0);
    const int blockSize = 256;
    int numBlocks = numStates/blockSize + (numStates % blockSize == 0 ? 0 : 1);
    initRandomStateBuffer<<<numBlocks, blockSize>>>(states, seed, numStates);
    hipDeviceSynchronize();
    printf("Seeding complete\n");
}

void OptixRenderer::initializeRandomStates()
{
    RTsize size[2];
    m_randomStatesBuffer->getSize(size[0], size[1]);
    int num = size[0]*size[1];
    initializeRandomStateBuffer(m_randomStatesBuffer, num);
}
