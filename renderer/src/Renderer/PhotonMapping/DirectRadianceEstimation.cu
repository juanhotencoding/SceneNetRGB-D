#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/Helpers/random.h"
#include "Renderer/Light.h"
#include "Renderer/RayType.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/ShadowPRD.h"
#include "Renderer/Helpers/light.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtBuffer<float3, 2> directRadianceBuffer;
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );
rtDeclareVariable(uint, localIterationNumber, , );

RT_PROGRAM void kernel()
{
    Hitpoint rec = raytracePassOutputBuffer[launchIndex];
    if (localIterationNumber == 0) {
        directRadianceBuffer[launchIndex] = optix::make_float3(0);
    }
    
    /*
    // Compute direct radiance
    */
    if (rec.flags != PRD_MISS) {
        int numLights = lights.size();
        const int numShadowSamples = NUM_SHADOW_SAMPLES; 
        float3 directRadiance = make_float3(0);
        if(numShadowSamples > 0)
        {
            float3 avgLightRadiance = make_float3(0.f);

            for(int shadowSample = 0; shadowSample < numShadowSamples; shadowSample++)
            {
                float sample = getRandomUniformFloat(&randomStates[launchIndex]);
                int randomLightIndex = intmin(int(sample*numLights), lights.size()-1);
                Light & light = lights[randomLightIndex];
                float scale = numLights;
                float3 lightContrib = getLightContribution(light, rec.position, rec.normal, sceneRootObject, randomStates[launchIndex]);
                avgLightRadiance += scale * lightContrib;
            }

            directRadiance = rec.attenuation*avgLightRadiance/numShadowSamples;
        }
        directRadianceBuffer[launchIndex] += directRadiance;
        directRadianceBuffer[launchIndex] += rec.radiance;
        //Ambient approximation for super fast non-GI rendering
        //directRadianceBuffer[launchIndex] += 0.04 * rec.attenuation;
    } else {
        directRadianceBuffer[launchIndex] += rec.radiance;
        //Ambient approximation for super fast non-GI rendering
        //directRadianceBuffer[launchIndex] += 0.04 * rec.attenuation;
    }
}

RT_PROGRAM void gatherAnyHitOnNonEmitter()
{
    shadowPrd.attenuation = 0.0f;
    rtTerminateRay();
}
