#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rendererConfig.h"
#include "Renderer/RadiancePRD.h"
#include "Renderer/Hitpoint.h"
#include "Renderer/RayType.h"
#include "Renderer/Helpers/random.h"
#include "Scene/SceneNetCamera.h"
#include "Util/helpers.h"

using namespace optix;

rtBuffer<unsigned int, 2> rawOutputDepthBuffer;
rtBuffer<uchar4, 2> outputDepthBuffer;
rtBuffer<unsigned int, 2> rawOutputVoxelBuffer;
rtBuffer<uchar4, 2> outputVoxelBuffer;
rtBuffer<unsigned int, 2> rawOutputClassBuffer;
rtBuffer<uchar4, 2> outputClassBuffer;
rtBuffer<unsigned int, 2> rawOutputInstanceBuffer;
rtBuffer<uchar4, 2> outputInstanceBuffer;
rtBuffer<float3, 2> hitpointBuffer;

rtDeclareVariable(unsigned int, MaxDepthInUnits, , );


rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(SceneNetCamera, camera, , );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

__device__ __inline uchar4 int_to_random_colour(const unsigned int value)
{
    return make_color(make_float3(static_cast<float>((value*2654435761)%213)/213.0,
                                  static_cast<float>((value*2654435761)%217)/217.0,
                                  static_cast<float>((value*2654435761)%231)/231.0));
}

__device__ __inline uchar4 depth_to_colour(const unsigned int depth)
{
    return make_color(make_float3(static_cast<float>(depth) / MaxDepthInUnits));
}


RT_PROGRAM void generateRay()
{
    RadiancePRD radiancePrd;
    float2 screen = make_float2(rawOutputDepthBuffer.size());
    float2 sample = make_float2(0.5,0.5);
    float2 d = ( make_float2(launchIndex) + sample ) / screen * 2.0f - 1.0f;
    float3 rayOrigin = camera.eye;
    float3 rayDirection = normalize(d.x*camera.camera_u + d.y*camera.camera_v + camera.lookdir);
    Ray ray(rayOrigin, rayDirection, RayType::GROUND_TRUTH, 0.001f);
    rtTrace(sceneRootObject, ray, radiancePrd);

    rawOutputDepthBuffer[launchIndex] = radiancePrd.gt_depth;
    rawOutputInstanceBuffer[launchIndex] = radiancePrd.instance_id;
    hitpointBuffer[launchIndex] = radiancePrd.position;
    
    outputDepthBuffer[launchIndex] = depth_to_colour(radiancePrd.gt_depth);
    outputInstanceBuffer[launchIndex] = int_to_random_colour(radiancePrd.instance_id);
}

//
// Miss program
//

RT_PROGRAM void miss()
{
    radiancePrd.gt_depth = 0;
    radiancePrd.instance_id = 0;
}

//
// Exception handler program
//

RT_PROGRAM void exception()
{
    printf("Exception GT!\n");
}
