#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <Util/helpers.h>

using namespace optix;

rtBuffer<float3, 2> rawOutputImageBuffer;
rtBuffer<uchar4, 2> outputImageBuffer;
rtBuffer<float3, 2> indirectRadianceBuffer;
rtBuffer<float3, 2> directRadianceBuffer;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(uint, resDownsample, , );
rtDeclareVariable(uint, numIterationsSqrt, , );

RT_PROGRAM void kernel()
{
    float3 finalRadiance = optix::make_float3(0,0,0);
    for (int i = 0; i < resDownsample; ++i) {
        for (int j = 0; j < resDownsample; ++j) {
            finalRadiance += directRadianceBuffer[launchIndex * resDownsample + optix::make_uint2(i,j)];
            //finalRadiance += directRadianceBuffer[launchIndex * resDownsample + optix::make_uint2(i,j)];
            finalRadiance += indirectRadianceBuffer[launchIndex * resDownsample + optix::make_uint2(i,j)];
        }
    }
    finalRadiance /= (resDownsample * resDownsample * numIterationsSqrt * numIterationsSqrt);
    rawOutputImageBuffer[launchIndex] = finalRadiance;
}
